#include "hip/hip_runtime.h"
#include "kernels.cuh"
#include <cub/warp/warp_reduce.cuh>

__global__ void me_p3(hipTextureObject_t texObj, float* Rx, float* rx, const int width, const int paddedWidth, const int height) 
{
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int localId = threadIdx.x;
    const int warpId = localId / 32;
	const int outputIndex = (y * paddedWidth) + x;

    __shared__ float RxLocal[64][36];
    __shared__ float rxLocal[8][64];
    __shared__ float rxSums[2][8];
    __shared__ typename hipcub::WarpReduce<float>::TempStorage tempStorage[2]; //2 warps

    //initialize shared memory with coalesced access
    for (int i = 0; i < 8; i++)
        rxLocal[i][localId] = 0.0f;
    if (x >= width) 
    {
        #pragma unroll
        for (int i = 0; i < 36; i++)
            RxLocal[localId][i] = 0.0f;
    }

    if (y >= height)
        return;

    if (x < width) 
    {
        int counter = 0;
        float x_[9];
        for (int j = x - 1; j <= x + 1; j++)
            for (int i = y - 1; i <= y + 1; i++)
                x_[counter++] = tex2D<float>(texObj, j, i);
        const float current_pixel = x_[4];

        //shift neighborhood values, so that consecutive values are neighbors only (to eliminate "if"s)
        #pragma unroll
        for (int i = 4; i < 8; i++)
            x_[i] = x_[i + 1];

        //calculate this thread's 36 local Rx and 8 local rx values
        counter = 0;
        #pragma unroll
        for (int i = 0; i < 8; i++) 
        {
            rxLocal[i][localId] = x_[i] * current_pixel;
            #pragma unroll
            for (int j = i; j < 8; j++)
                RxLocal[localId][counter++] = x_[i] * x_[j];
        }
    }

    //each thread will calculate the reduction sums of Rx and rx and write them to global memory
    //if image is padded we don't want to sum the garbage local array values, we could zero the local array
    //but it would cost time, instead it is better to calculate what is needed directly
    __syncthreads();
    float reduction_sum_Rx = 0.0f, reduction_sum_rx = 0.0f;
    #pragma unroll
    for (int j = 0; j < 64; j++)
        reduction_sum_Rx += RxLocal[j][RxMappings[localId]];
    Rx[outputIndex] = reduction_sum_Rx;

    //optimized summation for rx with cub
    float rxThreadSums[8];
    #pragma unroll
    for (int i = 0; i < 8; i++) 
        rxThreadSums[i] = hipcub::WarpReduce<float>(tempStorage[warpId]).Sum(rxLocal[i][localId]);
    __syncthreads();
    if (localId == 0 || localId == 32)
    {
        #pragma unroll
        for (int i = 0; i < 8; i++)
            rxSums[warpId][i] = rxThreadSums[i];
    }
    __syncthreads();
    if (localId < 8) 
         rx[(outputIndex / 8) + localId] = rxSums[0][localId] + rxSums[1][localId];

}

__global__ void calculate_neighbors_p3(hipTextureObject_t texObj, float* x_, const int width, const int height)
{
    const int x = blockIdx.y * blockDim.y + threadIdx.y;
    const int y = blockIdx.x * blockDim.x + threadIdx.x;
    const int outputIndex = (x * height + y);

    if (x < width && y < height) 
    {
        //store 8 neighboring pixels into global memory (coalesced writes)
        x_[0 * width * height + outputIndex] = tex2D<float>(texObj, x - 1, y - 1);
        x_[1 * width * height + outputIndex] = tex2D<float>(texObj, x - 1, y);
        x_[2 * width * height + outputIndex] = tex2D<float>(texObj, x - 1, y + 1);
        x_[3 * width * height + outputIndex] = tex2D<float>(texObj, x, y - 1);
        x_[4 * width * height + outputIndex] = tex2D<float>(texObj, x, y + 1);
        x_[5 * width * height + outputIndex] = tex2D<float>(texObj, x + 1, y - 1);
        x_[6 * width * height + outputIndex] = tex2D<float>(texObj, x + 1, y);
        x_[7 * width * height + outputIndex] = tex2D<float>(texObj, x + 1, y + 1);
    }
}