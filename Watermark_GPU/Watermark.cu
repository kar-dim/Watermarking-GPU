#include "hip/hip_runtime.h"
#include "cuda_utils.hpp"
#include "kernels.cuh"
#include "Watermark.cuh"
#include <af/hip/hip_runtime.h>
#include <arrayfire.h>
#include <cmath>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <memory>
#include <stdexcept>
#include <string>
#include <utility>

#define ME_MASK_CALCULATION_REQUIRED_NO false
#define ME_MASK_CALCULATION_REQUIRED_YES true

using std::string;
using std::cout;

//constructor without specifying input image yet, it must be supplied later by calling the appropriate public method
Watermark::Watermark(const string &w_file_path, const int p, const float psnr)
	:w_file_path(w_file_path), p(p), p_squared(p* p), p_squared_minus_one(p_squared - 1), p_squared_minus_one_squared(p_squared_minus_one* p_squared_minus_one), pad(p / 2), psnr(psnr) 
{
	rows = -1;
	cols = -1;
	af_cuda_stream = afcu::getStream(afcu::getNativeId(af::getDevice()));
	hipStreamCreate(&custom_kernels_stream);

}

//full constructor
Watermark::Watermark(const af::array &rgb_image, const af::array& image, const string &w_file_path, const int p, const float psnr)
	:Watermark::Watermark(w_file_path, p, psnr) 
{
	this->rgb_image = rgb_image;
	load_image(image);
	w = load_W(rows, cols);
}

//destructor, only custom kernels cuda stream must be destroyed
Watermark::~Watermark()
{
	hipStreamDestroy(custom_kernels_stream);
}

//supply the input image to apply watermarking and detection
void Watermark::load_image(const af::array& image) 
{
	this->image = image;
	rows = image.dims(0);
	cols = image.dims(1);
}

//helper method to load the random noise matrix W from the file specified.
af::array Watermark::load_W(const dim_t rows, const dim_t cols) const 
{
	std::ifstream w_stream(w_file_path.c_str(), std::ios::binary);
	if (!w_stream.is_open())
		throw std::runtime_error(string("Error opening '" + w_file_path + "' file for Random noise W array\n"));
	w_stream.seekg(0, std::ios::end);
	const auto total_bytes = w_stream.tellg();
	w_stream.seekg(0, std::ios::beg);
	if (rows * cols * sizeof(float) != total_bytes)
		throw std::runtime_error(string("Error: W file total elements != image dimensions! W file total elements: " + std::to_string(total_bytes / (sizeof(float))) + ", Image width: " + std::to_string(cols) + ", Image height: " + std::to_string(rows) + "\n"));
	std::unique_ptr<float> w_ptr(new float[rows * cols]);
	w_stream.read(reinterpret_cast<char*>(&w_ptr.get()[0]), total_bytes);
	return af::transpose(af::array(cols, rows, w_ptr.get()));
}

//helper method to copy an arrayfire cuda buffer into a cuda Texture Object Image (fast copy that happens in the device)
std::pair<hipTextureObject_t, hipArray*> Watermark::copy_array_to_texture_data(const af::array & array, const unsigned int rows, const unsigned int cols) const 
{
	hipArray* cuArray = cuda_utils::hipMallocArray(cols, rows);
	hipMemcpy2DToArray(cuArray, 0, 0, array.device<float>(), cols * sizeof(float), cols * sizeof(float), rows, hipMemcpyDeviceToDevice);
	hipResourceDesc resDesc = cuda_utils::createResourceDescriptor(cuArray);
	hipTextureDesc texDesc = cuda_utils::createTextureDescriptor();
	hipTextureObject_t texObj = cuda_utils::createTextureObject(resDesc, texDesc);
	return std::make_pair(texObj, cuArray);
}

//helper method for cleanup and to execute common tasks after the masking kernels are executed
void Watermark::synchronize_and_cleanup_texture_data(const std::pair<hipTextureObject_t, hipArray*> &texture_data, const af::array &array_to_unlock) const 
{
	hipDestroyTextureObject(texture_data.first);
	hipFreeArray(texture_data.second);
	array_to_unlock.unlock();
	hipStreamSynchronize(custom_kernels_stream);
}

//compute custom mask. supports simple kernels that just apply a mask per-pixel without needing any other configuration
af::array Watermark::compute_custom_mask(const af::array& image) const
{
	const auto rows = static_cast<unsigned int>(image.dims(0));
	const auto cols = static_cast<unsigned int>(image.dims(1));
	const af::array image_transpose = image.T();
	auto texture_data = copy_array_to_texture_data(image_transpose, rows, cols);
	float* mask_output = cuda_utils::cudaMallocPtr(rows * cols);
	auto dimensions = std::make_pair(cuda_utils::grid_size_calculate(dim3(32, 32), rows, cols), dim3(32, 32));
	nvf <<<dimensions.first, dimensions.second, 0, af_cuda_stream >>> (texture_data.first, mask_output, p*p, pad, cols, rows);
	synchronize_and_cleanup_texture_data(texture_data, image_transpose);
	return af::array(rows, cols, mask_output, afDevice);
}

//helper method to calculate the neighbors ("x_" array)
af::array Watermark::calculate_neighbors_array(const af::array& array, const int p, const int p_squared, const int pad) const 
{
	af::array array_unwrapped = af::unwrap(array, p, p, 1, 1, pad, pad, false);
	return af::join(1, array_unwrapped(af::span, af::seq(0, (p_squared / 2) - 1)), array_unwrapped(af::span, af::seq((p_squared / 2) + 1, af::end)));
}

//helper method to sum the incomplete Rx_partial and rx_partial arrays which were produced from the custom kernel
//and to transform them to the correct size, so that they can be used by the system solver
std::pair<af::array, af::array> Watermark::correlation_arrays_transformation(const af::array& Rx_partial, const af::array& rx_partial, const int padded_cols) const
{
	af::array Rx_partial_sums = af::moddims(Rx_partial, p_squared_minus_one_squared, (padded_cols * rows) / p_squared_minus_one_squared);
	af::array rx_partial_sums = af::moddims(rx_partial, p_squared_minus_one, (padded_cols * rows) / p_squared_minus_one);
	//reduction sum of blocks
	//all [p^2-1,1] blocks will be summed in rx
	//all [p^2-1, p^2-1] blocks will be summed in Rx
	af::array Rx = af::moddims(af::sum(Rx_partial_sums, 1), p_squared_minus_one, p_squared_minus_one);
	af::array rx = af::sum(rx_partial_sums, 1);
	return std::make_pair(Rx, rx);
}

//Main watermark embedding method
af::array Watermark::make_and_add_watermark(af::array& coefficients, float& a, MASK_TYPE mask_type, IMAGE_TYPE image_type) const
{
	af::array mask, error_sequence;
	mask = mask_type == MASK_TYPE::ME ? 
		compute_prediction_error_mask(image, error_sequence, coefficients, ME_MASK_CALCULATION_REQUIRED_YES) :
		compute_custom_mask(image);
	const af::array u = mask * w;
	const float divisor = std::sqrt(af::sum<float>(af::pow(u, 2)) / (image.elements()));
	a = (255.0f / std::sqrt(std::pow(10.0f, psnr / 10.0f))) / divisor;
	return image_type == IMAGE_TYPE::RGB ?
		af::clamp(rgb_image + af::tile((u * a), 1, 1, static_cast<unsigned int>(image.dims(2))), 0, 255) :
		af::clamp(image + (u * a), 0, 255);
}

//Compute prediction error mask. Used in both creation and detection of the watermark.
//can also calculate error sequence and prediction error filter
af::array Watermark::compute_prediction_error_mask(const af::array& image, af::array& error_sequence, af::array& coefficients, const bool mask_needed) const
{
	const auto rows = static_cast<unsigned int>(image.dims(0));
	const auto cols = static_cast<unsigned int>(image.dims(1));
	const af::array image_transpose = image.T();
	const auto padded_cols = (cols % 64 == 0) ? cols : cols + 64 - (cols % 64);
	//copy image to texture cache and call custom kernel
	auto texture_data = copy_array_to_texture_data(image_transpose, rows, cols);
	float* Rx_buff = cuda_utils::cudaMallocPtr(rows * padded_cols);
	float* rx_buff = cuda_utils::cudaMallocPtr(rows * padded_cols);
	auto dimensions = std::make_pair(cuda_utils::grid_size_calculate(dim3(1, 64), rows, padded_cols), dim3(1, 64));
	me_p3 <<<dimensions.first, dimensions.second, 0, custom_kernels_stream>>> (texture_data.first, Rx_buff, rx_buff, cols, padded_cols, rows);
	af::array x_ = calculate_neighbors_array(image, p, p_squared, pad);
	//cleanup and calculation of coefficients, error sequence and mask
	synchronize_and_cleanup_texture_data(texture_data, image_transpose);
	const auto correlation_arrays = correlation_arrays_transformation(af::array(padded_cols, rows, Rx_buff, afDevice), af::array(padded_cols, rows, rx_buff, afDevice), padded_cols);
	coefficients = af::solve(correlation_arrays.first, correlation_arrays.second);
	error_sequence = af::moddims(af::flat(image).T() - af::matmulTT(coefficients, x_), rows, cols);
	if (mask_needed) {
		const af::array error_sequence_abs = af::abs(error_sequence);
		return error_sequence_abs / af::max<float>(error_sequence_abs);
	}
	return af::array();
}

//helper method that calculates the error sequence by using a supplied prediction filter coefficients
af::array Watermark::calculate_error_sequence(const af::array& u, const af::array& coefficients) const 
{
	return af::moddims(af::flat(u).T() - af::matmulTT(coefficients, calculate_neighbors_array(u, p, p_squared, pad)), u.dims(0), u.dims(1));
}

//overloaded, fast mask calculation by using a supplied prediction filter
af::array Watermark::compute_prediction_error_mask(const af::array& image, const af::array& coeficcients, af::array& error_sequence) const
{
	error_sequence = calculate_error_sequence(image, coeficcients);
	const af::array error_sequence_abs = af::abs(error_sequence);
	return error_sequence_abs / af::max<float>(error_sequence_abs);
}

//helper method used in detectors
float Watermark::calculate_correlation(const af::array& e_u, const af::array& e_z) const 
{
	double dot_ez_eu = af::dot<double>(af::flat(e_u), af::flat(e_z)); //dot() needs vectors, so we flatten the arrays
	return static_cast<float>(dot_ez_eu / (af::norm(e_z) * af::norm(e_u)));
}

//the main mask detector function
float Watermark::mask_detector(const af::array& watermarked_image, MASK_TYPE mask_type) const
{
	af::array mask, e_z, a_z;
	if (mask_type == MASK_TYPE::NVF) {
		compute_prediction_error_mask(watermarked_image, e_z, a_z, ME_MASK_CALCULATION_REQUIRED_NO);
		mask = compute_custom_mask(watermarked_image);
	}
	else
		mask = compute_prediction_error_mask(watermarked_image, e_z, a_z, ME_MASK_CALCULATION_REQUIRED_YES);
	const af::array u = mask * w;
	const af::array e_u = calculate_error_sequence(u, a_z);
	return calculate_correlation(e_u, e_z);
}

//fast mask detector, used only for a video frame, by detecting the watermark based on previous frame (coefficients, x_ are supplied)
float Watermark::mask_detector_prediction_error_fast(const af::array& watermarked_image, const af::array& coefficients) const
{
	af::array m_e, e_z, e_u, a_u;
	m_e = compute_prediction_error_mask(watermarked_image, coefficients, e_z);
	const af::array u = m_e * w;
	compute_prediction_error_mask(u, e_u, a_u, ME_MASK_CALCULATION_REQUIRED_NO);
	return calculate_correlation(e_u, e_z);
}

//helper method to display an af::array in a window
void Watermark::display_array(const af::array& array, const int width, const int height) 
{
	af::Window window(width, height);
	while (!window.close())
		window.image(array);
}