#include "hip/hip_runtime.h"
#include "Watermark.cuh"
#include "cuda_utils.hpp"
#include "kernels.cuh"
#include <af/hip/hip_runtime.h>
#include <arrayfire.h>
#include <cmath>
#include <hip/hip_runtime.h>
#include <fstream>
#include <memory>
#include <stdexcept>
#include <string>
#include <utility>

#define ME_MASK_CALCULATION_REQUIRED_NO false
#define ME_MASK_CALCULATION_REQUIRED_YES true

using std::string;

//constructor without specifying input image yet, it must be supplied later by calling the appropriate public method
Watermark::Watermark(const string &w_file_path, const int p, const float psnr)
	:w_file_path(w_file_path), p(p), strength_factor((255.0f / sqrt(pow(10.0f, psnr / 10.0f))))
{
	if (p != 3 && p != 5 && p != 7 && p != 9)
		throw std::runtime_error(string("Wrong p parameter: ") + std::to_string(p) + "!\n");
	rows = -1;
	cols = -1;
	af_cuda_stream = afcu::getStream(afcu::getNativeId(af::getDevice()));
	hipStreamCreate(&custom_kernels_stream);

}

//full constructor
Watermark::Watermark(const af::array &rgb_image, const af::array& image, const string &w_file_path, const int p, const float psnr)
	:Watermark::Watermark(w_file_path, p, psnr) 
{
	this->rgb_image = rgb_image;
	load_image(image);
	load_W(rows, cols);
}

//destructor, only custom kernels cuda stream must be destroyed
Watermark::~Watermark()
{
	hipStreamDestroy(custom_kernels_stream);
}

//supply the input image to apply watermarking and detection
void Watermark::load_image(const af::array& image) 
{
	this->image = image;
	rows = image.dims(0);
	cols = image.dims(1);
}

//helper method to load the random noise matrix W from the file specified.
void Watermark::load_W(const dim_t rows, const dim_t cols) 
{
	std::ifstream w_stream(w_file_path.c_str(), std::ios::binary);
	if (!w_stream.is_open())
		throw std::runtime_error(string("Error opening '" + w_file_path + "' file for Random noise W array\n"));
	w_stream.seekg(0, std::ios::end);
	const auto total_bytes = w_stream.tellg();
	w_stream.seekg(0, std::ios::beg);
	if (rows * cols * sizeof(float) != total_bytes)
		throw std::runtime_error(string("Error: W file total elements != image dimensions! W file total elements: " + std::to_string(total_bytes / (sizeof(float))) + ", Image width: " + std::to_string(cols) + ", Image height: " + std::to_string(rows) + "\n"));
	std::unique_ptr<float> w_ptr(new float[rows * cols]);
	w_stream.read(reinterpret_cast<char*>(&w_ptr.get()[0]), total_bytes);
	this->w = af::transpose(af::array(cols, rows, w_ptr.get()));
}

//compute custom mask. supports simple kernels that just apply a mask per-pixel without needing any other configuration
af::array Watermark::compute_custom_mask(const af::array& image) const
{
	const auto rows = static_cast<unsigned int>(image.dims(0));
	const auto cols = static_cast<unsigned int>(image.dims(1));
	const af::array image_transpose = image.T();
	const auto texture_data = cuda_utils::copyArrayToTexture(image_transpose.device<float>(), rows, cols);
	float* mask_output = cuda_utils::cudaMallocPtr(rows * cols);
	const auto dimensions = std::make_pair(cuda_utils::gridSizeCalculate(dim3(16, 16), rows, cols), dim3(16, 16));
	switch (p) {
		case 3: nvf<3> <<<dimensions.first, dimensions.second, 0, custom_kernels_stream >>> (texture_data.first, mask_output, cols, rows); break;
		case 5: nvf<5> <<<dimensions.first, dimensions.second, 0, custom_kernels_stream >>> (texture_data.first, mask_output, cols, rows); break;
		case 7: nvf<7> <<<dimensions.first, dimensions.second, 0, custom_kernels_stream >>> (texture_data.first, mask_output, cols, rows); break;
		case 9: nvf<9> <<<dimensions.first, dimensions.second, 0, custom_kernels_stream >>> (texture_data.first, mask_output, cols, rows); break;
	}
	cuda_utils::synchronizeAndCleanupTexture(custom_kernels_stream, texture_data);
	image_transpose.unlock();
	return af::array(rows, cols, mask_output, afDevice);
}

//helper method to calculate the neighbors ("x_" array)
af::array Watermark::calculate_neighbors_array(const af::array& array) const 
{
	const int center = (p * p) / 2;
	af::array unwrapped = af::unwrap(array, p, p, 1, 1, p / 2, p / 2, false);
	return af::join(1, unwrapped(af::span, af::seq(0, center - 1)), unwrapped(af::span, af::seq(center + 1, af::end)));
}

//helper method to sum the incomplete Rx_partial and rx_partial arrays which were produced from the custom kernel
//and to transform them to the correct size, so that they can be used by the system solver
std::pair<af::array, af::array> Watermark::correlation_arrays_transformation(const af::array& Rx_partial, const af::array& rx_partial, const int rows, const int padded_cols) const
{
	const int p_sq_minus_one = (p * p) - 1;
	const int p_sq_minus_one_sq = p_sq_minus_one * p_sq_minus_one;
	//reduction sum of blocks
	//all [p^2-1,1] blocks will be summed in rx
	//all [p^2-1, p^2-1] blocks will be summed in Rx
	const af::array Rx = af::moddims(af::sum(af::moddims(Rx_partial, p_sq_minus_one_sq, (padded_cols * rows) / p_sq_minus_one_sq), 1), p_sq_minus_one, p_sq_minus_one);
	const af::array rx = af::sum(af::moddims(rx_partial, p_sq_minus_one, (padded_cols * rows) / (8 * p_sq_minus_one)), 1);
	return std::make_pair(Rx, rx);
}

//Main watermark embedding method
af::array Watermark::make_and_add_watermark(af::array& coefficients, float& a, MASK_TYPE mask_type, IMAGE_TYPE type) const
{
	af::array error_sequence;
	const af::array mask = mask_type == MASK_TYPE::ME ?
		compute_prediction_error_mask(image, error_sequence, coefficients, ME_MASK_CALCULATION_REQUIRED_YES) :
		compute_custom_mask(image);
	const af::array u = mask * w;
	a = strength_factor / sqrt(af::sum<float>(af::pow(u, 2)) / image.elements());
	return af::clamp((type == IMAGE_TYPE::RGB ? rgb_image : image) + (u * a), 0, 255);
}

//Compute prediction error mask. Used in both creation and detection of the watermark.
//can also calculate error sequence and prediction error filter
af::array Watermark::compute_prediction_error_mask(const af::array& image, af::array& error_sequence, af::array& coefficients, const bool mask_needed) const
{
	const auto rows = static_cast<unsigned int>(image.dims(0));
	const auto cols = static_cast<unsigned int>(image.dims(1));
	const af::array image_transpose = image.T();
	const auto padded_cols = (cols % 64 == 0) ? cols : cols + 64 - (cols % 64);
	//enqueue "x_" kernel (which is heavy)
	const af::array x_ = calculate_neighbors_array(image);
	//initialize custom kernel memory
	float* Rx_buff = cuda_utils::cudaMallocPtr(rows * padded_cols);
	float* rx_buff = cuda_utils::cudaMallocPtr(rows * padded_cols / 8);
	//do a texture copy (for custom kernel)
	const auto texture_data = cuda_utils::copyArrayToTexture(image_transpose.device<float>(), rows, cols);
	const auto dimensions = std::make_pair(cuda_utils::gridSizeCalculate(dim3(1, 64), rows, padded_cols), dim3(1, 64));
	me_p3 <<<dimensions.first, dimensions.second, 0, custom_kernels_stream>>> (texture_data.first, Rx_buff, rx_buff, cols, padded_cols, rows);
	//cleanup and calculation of coefficients, error sequence and mask
	cuda_utils::synchronizeAndCleanupTexture(custom_kernels_stream, texture_data);
	image_transpose.unlock();
	const auto correlation_arrays = correlation_arrays_transformation(af::array(padded_cols, rows, Rx_buff, afDevice), af::array(padded_cols / 8, rows, rx_buff, afDevice), rows, padded_cols);
	coefficients = af::solve(correlation_arrays.first, correlation_arrays.second);
	error_sequence = af::moddims(af::flat(image).T() - af::matmulTT(coefficients, x_), rows, cols);
	if (mask_needed) {
		const af::array error_sequence_abs = af::abs(error_sequence);
		return error_sequence_abs / af::max<float>(error_sequence_abs);
	}
	return af::array();
}

//helper method that calculates the error sequence by using a supplied prediction filter coefficients
af::array Watermark::calculate_error_sequence(const af::array& u, const af::array& coefficients) const 
{
	return af::moddims(af::flat(u).T() - af::matmulTT(coefficients, calculate_neighbors_array(u)), u.dims(0), u.dims(1));
}

//overloaded, fast mask calculation by using a supplied prediction filter
af::array Watermark::compute_prediction_error_mask(const af::array& image, const af::array& coefficients, af::array& error_sequence) const
{
	error_sequence = calculate_error_sequence(image, coefficients);
	const af::array error_sequence_abs = af::abs(error_sequence);
	return error_sequence_abs / af::max<float>(error_sequence_abs);
}

//helper method used in detectors
float Watermark::calculate_correlation(const af::array& e_u, const af::array& e_z) const 
{
	return af::dot<float>(af::flat(e_u), af::flat(e_z)) / static_cast<float>(af::norm(e_z) * af::norm(e_u));
}

//the main mask detector function
float Watermark::mask_detector(const af::array& watermarked_image, MASK_TYPE mask_type) const
{
	af::array mask, e_z, a_z;
	if (mask_type == MASK_TYPE::NVF) {
		compute_prediction_error_mask(watermarked_image, e_z, a_z, ME_MASK_CALCULATION_REQUIRED_NO);
		mask = compute_custom_mask(watermarked_image);
	}
	else
		mask = compute_prediction_error_mask(watermarked_image, e_z, a_z, ME_MASK_CALCULATION_REQUIRED_YES);
	const af::array u = mask * w;
	const af::array e_u = calculate_error_sequence(u, a_z);
	return calculate_correlation(e_u, e_z);
}

//fast mask detector, used only for a video frame, by detecting the watermark based on previous frame (coefficients, x_ are supplied)
float Watermark::mask_detector_prediction_error_fast(const af::array& watermarked_image, const af::array& coefficients) const
{
	af::array e_z, e_u, a_u;
	const af::array m_e = compute_prediction_error_mask(watermarked_image, coefficients, e_z);
	const af::array u = m_e * w;
	compute_prediction_error_mask(u, e_u, a_u, ME_MASK_CALCULATION_REQUIRED_NO);
	return calculate_correlation(e_u, e_z);
}

//helper method to display an af::array in a window
void Watermark::display_array(const af::array& array, const int width, const int height) 
{
	af::Window window(width, height);
	while (!window.close())
		window.image(array);
}