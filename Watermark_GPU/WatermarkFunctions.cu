#include "hip/hip_runtime.h"
#include "kernels.cuh"
#include "WatermarkFunctions.cuh"
#include <fstream>
#include <arrayfire.h>
#include <iostream>
#include <string>
#include <af/hip/hip_runtime.h>
#include <cmath>
#include <memory>
#include <functional>
#include <hip/hip_runtime.h>
#include "cuda_utils.h"

using std::cout;

//constructor without specifying input image yet, it must be supplied later by calling the appropriate public method
WatermarkFunctions::WatermarkFunctions(const std::string w_file_path, const int p, const float psnr)
	:p(p), p_squared(p* p), p_squared_minus_one(p_squared - 1), p_squared_minus_one_squared(p_squared_minus_one* p_squared_minus_one), pad(p / 2), psnr(psnr), w_file_path(w_file_path) {
	this->af_cuda_stream = afcu::getStream(afcu::getNativeId(af::getDevice()));
	hipStreamCreate(&custom_kernels_stream);
	this->rows = -1;
	this->cols = -1;
}

WatermarkFunctions::~WatermarkFunctions()
{
	hipStreamDestroy(custom_kernels_stream);
}

//full constructor
WatermarkFunctions::WatermarkFunctions(const af::array& image, const std::string w_file_path, const int p, const float psnr)
	:WatermarkFunctions::WatermarkFunctions(w_file_path, p, psnr) {
	load_image(image);
	load_W(this->rows, this->cols);
}

//supply the input image to apply watermarking and detection
void WatermarkFunctions::load_image(const af::array& image) {
	this->image = image;
	this->rows = image.dims(0);
	this->cols = image.dims(1);
}

//helper method to load the random noise matrix W from the file specified.
void WatermarkFunctions::load_W(const dim_t rows, const dim_t cols) {
	std::ifstream w_stream(this->w_file_path.c_str(), std::ios::binary);
	if (!w_stream.is_open()) {
		std::string error_str("Error opening '" + this->w_file_path + "' file for Random noise W array");
		throw std::exception(error_str.c_str());
	}
	w_stream.seekg(0, std::ios::end);
	const auto total_bytes = w_stream.tellg();
	w_stream.seekg(0, std::ios::beg);
	if (rows * cols * sizeof(float) != total_bytes) {
		std::string error_str("Error: W file total elements != image dimensions! W file total elements: " + std::to_string(total_bytes / (sizeof(float))) + std::string(", Image width: ") + std::to_string(cols) + std::string(", Image height: ") + std::to_string(rows));
		throw std::exception(error_str.c_str());
	}
	std::unique_ptr<float> w_ptr(new float[rows * cols]);
	w_stream.read(reinterpret_cast<char*>(&w_ptr.get()[0]), total_bytes);
	this->w = af::transpose(af::array(cols, rows, w_ptr.get()));
}

//helper method to copy an arrayfire cuda buffer into a cuda Texture Object Image (fast copy that happens in the device)
std::pair<hipTextureObject_t, hipArray*> WatermarkFunctions::copy_array_to_texture_data(const af::array & array, const unsigned int rows, const unsigned int cols) {
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	hipArray* cuArray;
	hipMallocArray(&cuArray, &channelDesc, cols, rows);
	hipMemcpy2DToArray(cuArray, 0, 0, array.device<float>(), cols * sizeof(float), cols * sizeof(float), rows, hipMemcpyDeviceToDevice);
	struct hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = cuArray;
	struct hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.addressMode[0] = hipAddressModeBorder;
	texDesc.addressMode[1] = hipAddressModeBorder;
	texDesc.filterMode = hipFilterModePoint;
	texDesc.readMode = hipReadModeElementType;
	texDesc.normalizedCoords = 0;
	hipTextureObject_t texObj = 0;
	hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);
	return std::make_pair(texObj, cuArray);
}

//helper method for cleanup and to execute common tasks after the masking kernels are executed
void WatermarkFunctions::synchronize_and_cleanup_texture_data(const std::pair<hipTextureObject_t, hipArray*> &texture_data, const af::array &array_to_unlock) {
	hipDeviceSynchronize();
	hipDestroyTextureObject(texture_data.first);
	hipFreeArray(texture_data.second);
	array_to_unlock.unlock();
}

//compute custom mask. supports simple kernels that just apply a mask per-pixel without needing any other configuration
void WatermarkFunctions::compute_custom_mask(const af::array& image, af::array& m)
{
	const auto rows = static_cast<unsigned int>(image.dims(0));
	const auto cols = static_cast<unsigned int>(image.dims(1));
	const af::array image_transpose = image.T();
	auto texture_data = copy_array_to_texture_data(image_transpose, rows, cols);
	float* mask_output = cuda_utils::cudaMallocPtr<float>(rows * cols);
	auto dimensions = std::make_pair(cuda_utils::grid_size_calculate(dim3(16, 16), rows, cols), dim3(16, 16));
	nvf <<<dimensions.first, dimensions.second, 0, af_cuda_stream >>> (texture_data.first, mask_output, p*p, pad, cols, rows);
	synchronize_and_cleanup_texture_data(texture_data, image_transpose);
	m = af::array(rows, cols, mask_output, afDevice);
}

//helper method to calculate the neighbors ("x_" array)
af::array WatermarkFunctions::calculate_neighbors_array(const af::array& array, const int p, const int p_squared, const int pad) {
	af::array array_unwrapped = af::unwrap(array, p, p, 1, 1, pad, pad, false);
	return af::join(1, array_unwrapped(af::span, af::seq(0, (p_squared / 2) - 1)), array_unwrapped(af::span, af::seq((p_squared / 2) + 1, af::end)));
}

//helper method to sum the incomplete Rx_partial and rx_partial arrays which were produced from the custom kernel
//and to transform them to the correct size, so that they can be used by the system solver
std::pair<af::array, af::array> WatermarkFunctions::correlation_arrays_transformation(const af::array& Rx_partial, const af::array& rx_partial, const int padded_cols) {
	af::array Rx_partial_sums = af::moddims(Rx_partial, p_squared_minus_one_squared, (padded_cols * rows) / p_squared_minus_one_squared);
	af::array rx_partial_sums = af::moddims(rx_partial, p_squared_minus_one, (padded_cols * rows) / p_squared_minus_one);
	//reduction sum of blocks
	//all [p^2-1,1] blocks will be summed in rx
	//all [p^2-1, p^2-1] blocks will be summed in Rx
	af::array Rx = af::moddims(af::sum(Rx_partial_sums, 1), p_squared_minus_one, p_squared_minus_one);
	af::array rx = af::sum(rx_partial_sums, 1);
	return std::make_pair(Rx, rx);
}

af::array WatermarkFunctions::make_and_add_watermark(float& a, const std::function<void(const af::array&, af::array&, af::array&)>& compute_mask)
{
	af::array m, error_sequence;
	compute_mask(image, m, error_sequence);
	const af::array u = m * w;
	const float divisor = std::sqrt(af::sum<float>(af::pow(u, 2)) / (image.elements()));
	a = (255.0f / std::sqrt(std::pow(10.0f, psnr / 10.0f))) / divisor;
	return image + (a * u);
}

//public method called from host to apply the custom mask and return the watermarked image
af::array WatermarkFunctions::make_and_add_watermark_custom(float& a)
{
	return make_and_add_watermark(a, [&](const af::array& image, af::array& m, af::array& error_sequence) {
		compute_custom_mask(image, m);
	});
}

//public method called from host to apply the prediction error mask and return the watermarked image
af::array WatermarkFunctions::make_and_add_watermark_prediction_error(af::array& coefficients, float& a)
{
	return make_and_add_watermark(a, [&](const af::array& image, af::array& m, af::array& error_sequence) {
		compute_prediction_error_mask(image, m, error_sequence, coefficients, ME_MASK_CALCULATION_REQUIRED_YES);
	});
}

//Compute prediction error mask. Used in both creation and detection of the watermark.
//can also calculate error sequence and prediction error filter
void WatermarkFunctions::compute_prediction_error_mask(const af::array& image, af::array& m_e, af::array& error_sequence, af::array& coefficients, const bool mask_needed)
{
	const auto rows = static_cast<unsigned int>(image.dims(0));
	const auto cols = static_cast<unsigned int>(image.dims(1));
	const af::array image_transpose = image.T();
	const auto padded_cols = (cols % 64 == 0) ? cols : cols + 64 - (cols % 64);
	//copy arrayfire array from device to device's texture cache and allocate Rx,rx buffers
	auto texture_data = copy_array_to_texture_data(image_transpose, rows, cols);
	float* Rx_buff = cuda_utils::cudaMallocPtr<float>(rows * padded_cols);
	float* rx_buff = cuda_utils::cudaMallocPtr<float>(rows * padded_cols);
	//call custom kernel to fill Rx and rx partial sums (in different stream than arrayfire, may help)
	auto dimensions = std::make_pair(cuda_utils::grid_size_calculate(dim3(1, 64), rows, padded_cols), dim3(1, 64));
	me_p3 <<<dimensions.first, dimensions.second, 0, custom_kernels_stream>>> (texture_data.first, Rx_buff, rx_buff, cols, padded_cols, rows);
	//calculate the neighbors "x_" array
	af::array x_ = calculate_neighbors_array(image, p, p_squared, pad);
	//wait for custom kernel to finish and release texture memory
	synchronize_and_cleanup_texture_data(texture_data, image_transpose);
	//transform the partial Rx,rx arrays by summing and changing their dimensions
	const auto correlation_arrays = correlation_arrays_transformation(af::array(padded_cols, rows, Rx_buff, afDevice), af::array(padded_cols, rows, rx_buff, afDevice), padded_cols);
	//solve the system to get coefficients and error sequence, and optionally the mask if needed
	coefficients = af::solve(correlation_arrays.first, correlation_arrays.second);
	error_sequence = af::moddims(af::flat(image).T() - af::matmulTT(coefficients, x_), rows, cols);
	if (mask_needed) {
		const af::array error_sequence_abs = af::abs(error_sequence);
		m_e = error_sequence_abs / af::max<float>(error_sequence_abs);
	}
}

//helper method that calculates the error sequence by using a supplied prediction filter coefficients
af::array WatermarkFunctions::calculate_error_sequence(const af::array& u, const af::array& coefficients) {
	return af::moddims(af::flat(u).T() - af::matmulTT(coefficients, calculate_neighbors_array(u, p, p_squared, pad)), u.dims(0), u.dims(1));
}

//overloaded, fast mask calculation by using a supplied prediction filter
void WatermarkFunctions::compute_prediction_error_mask(const af::array& image, const af::array& coeficcients, af::array& m_e, af::array& error_sequence)
{
	error_sequence = calculate_error_sequence(image, coeficcients);
	const af::array error_sequence_abs = af::abs(error_sequence);
	m_e = error_sequence_abs / af::max<float>(error_sequence_abs);
}

//fast prediction error sequence calculation by using a supplied prediction filter (calls helper method)
af::array WatermarkFunctions::compute_error_sequence(const af::array& u, const af::array& coefficients)
{
	return calculate_error_sequence(u, coefficients);
}

//helper method used in detectors
float WatermarkFunctions::calculate_correlation(const af::array& e_u, const af::array& e_z) {
	float dot_ez_eu = af::dot<float>(af::flat(e_u), af::flat(e_z)); //dot() needs vectors, so we flatten the arrays
	float d_ez = std::sqrt(af::sum<float>(af::pow(e_z, 2)));
	float d_eu = std::sqrt(af::sum<float>(af::pow(e_u, 2)));
	return dot_ez_eu / (d_ez * d_eu);
}

//the main mask detector function
float WatermarkFunctions::mask_detector(const af::array& image, bool custom_mask)
{
	af::array m, e_z, a_z;
	if (custom_mask == CUSTOM_MASK_CALCULATION_REQUIRED_YES) {
		compute_prediction_error_mask(image, m, e_z, a_z, ME_MASK_CALCULATION_REQUIRED_NO);
		compute_custom_mask(image, m);
	}
	else {
		compute_prediction_error_mask(image, m, e_z, a_z, ME_MASK_CALCULATION_REQUIRED_YES);
	}
	const af::array u = m * w;
	const af::array e_u = compute_error_sequence(u, a_z);
	return calculate_correlation(e_u, e_z);
}

//fast mask detector, used only for a video frame, by detecting the watermark based on previous frame (coefficients, x_ are supplied)
float WatermarkFunctions::mask_detector_prediction_error_fast(const af::array& watermarked_image, const af::array& coefficients)
{
	af::array m_e, e_z, m_eu, e_u, a_u;
	compute_prediction_error_mask(watermarked_image, coefficients, m_e, e_z);
	const af::array u = m_e * w;
	compute_prediction_error_mask(u, m_eu, e_u, a_u, ME_MASK_CALCULATION_REQUIRED_NO);
	return calculate_correlation(e_u, e_z);
}

//calls main mask detector for custom masks
float WatermarkFunctions::mask_detector_custom(const af::array& watermarked_image) {
	return mask_detector(watermarked_image, CUSTOM_MASK_CALCULATION_REQUIRED_YES);
}

//calls main mask detector for prediction error mask
float WatermarkFunctions::mask_detector_prediction_error(const af::array& watermarked_image) {
	return mask_detector(watermarked_image, CUSTOM_MASK_CALCULATION_REQUIRED_NO);
}

//helper method to display an af::array in a window
void WatermarkFunctions::display_array(const af::array& array, const int width, const int height) {
	af::Window window(width, height);
	while (!window.close())
		window.image(array);
}